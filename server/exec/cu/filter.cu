#include "hip/hip_runtime.h"
/*nvcc filter.cu `pkg-config --cflags --libs opencv`*/

#include <stdlib.h>
#include <string>
#include <opencv/highgui.h>
//#include "utils/cheader.h"

#define BLUR_WINDOW 15

typedef enum color {BLUE, GREEN, RED} Color;

__global__ void blur(unsigned char *src, unsigned char *dest, int width, int heigth, int blur_window, int step, int channels) {
	int i, j, side_pixels, cells;
	int ren, col, tmp_ren, tmp_col;
	float r, g, b;

	ren = blockIdx.x;
	col = threadIdx.x;
	side_pixels = (blur_window - 1) / 2;
	cells = (blur_window * blur_window);
	r = 0; g = 0; b = 0;
	for (i = -side_pixels; i <= side_pixels; i++) {
		for (j = -side_pixels; j <= side_pixels; j++) {
			tmp_ren = MIN( MAX(ren + i, 0), heigth - 1 );
			tmp_col = MIN( MAX(col + j, 0), width - 1);

			r += (float) src[(tmp_ren * step) + (tmp_col * channels) + RED];
			g += (float) src[(tmp_ren * step) + (tmp_col * channels) + GREEN];
			b += (float) src[(tmp_ren * step) + (tmp_col * channels) + BLUE];
		}
	}

	dest[(ren * step) + (col * channels) + RED] =  (unsigned char) (r / cells);
	dest[(ren * step) + (col * channels) + GREEN] = (unsigned char) (g / cells);
	dest[(ren * step) + (col * channels) + BLUE] = (unsigned char) (b / cells);
}

__global__ void gray(unsigned char *src, unsigned char *dest, int width, int height, int step, int channels){
  int ren, col;
	float r, g, b, avg;

	ren = blockIdx.x;
	col = threadIdx.x;
	r = 0; g = 0; b = 0;

	r = (float) src[(ren * step) + (col * channels) + RED];
	g = (float) src[(ren * step) + (col * channels) + GREEN];
	b = (float) src[(ren * step) + (col * channels) + BLUE];

  avg = (r + g + b)/3.0;

	dest[(ren * step) + (col * channels) + RED] =  (unsigned char) (avg);
	dest[(ren * step) + (col * channels) + GREEN] = (unsigned char) (avg);
	dest[(ren * step) + (col * channels) + BLUE] = (unsigned char) (avg);
}

__global__ void edge(unsigned char *src, unsigned char *dest, int width, int height, int step, int channels){

}

int main(int argc, char* argv[]) {
	int step, size;
	unsigned char *dev_src, *dev_dest;

	char src_name[255];
	char filter_t[255];
	char dest_name[255];
  char dir[255] = "img/";

	strcpy(src_name, argv[1]);
	strcpy(filter_t, argv[2]);
	strcpy(dest_name, argv[3]);

	IplImage *src = cvLoadImage(src_name, CV_LOAD_IMAGE_COLOR);
	IplImage *dest = cvCreateImage(cvSize(src->width, src->height), IPL_DEPTH_8U, 3);

	size = src->width * src->height * src->nChannels * sizeof(uchar);
	step = src->widthStep / sizeof(uchar);

	hipMalloc((void**) &dev_src, size);
	hipMalloc((void**) &dev_dest, size);

	hipMemcpy(dev_src, src->imageData, size, hipMemcpyHostToDevice);

  if(compare(filter_t,"blur") == 0){
    blur<<<src->height, src->width>>>(dev_src, dev_dest, src->width, src->height, BLUR_WINDOW, step, src->nChannels);
  }

  if(compare(filter_t,"gray") == 0){
    gray<<<src->height, src->width>>>(dev_src, dev_dest, src->width, src->height, step, src->nChannels);
  }

	hipMemcpy(dest->imageData, dev_dest, size, hipMemcpyDeviceToHost);

	cvSaveImage(strcat(dir,dest_name) , dest);

	hipFree(dev_dest);
	hipFree(dev_src);

	return 0;
}
