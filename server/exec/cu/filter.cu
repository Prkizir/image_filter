#include "hip/hip_runtime.h"
/*nvcc filter.cu `pkg-config --cflags --libs opencv`*/

#include <stdlib.h>
#include <string>
#include <opencv/highgui.h>
//#include "utils/cheader.h"

#define BLUR_WINDOW 15

typedef enum color {BLUE, GREEN, RED} Color;

__global__ void gray(unsigned char *src, unsigned char *dest, int width, int height, int step, int channels){
  int ren, col;
	float r, g, b, avg;

	ren = blockIdx.x;
	col = threadIdx.x;
	r = 0; g = 0; b = 0;

	r = (float) src[(ren * step) + (col * channels) + RED];
	g = (float) src[(ren * step) + (col * channels) + GREEN];
	b = (float) src[(ren * step) + (col * channels) + BLUE];

  avg = (r + g + b)/3.0;

	dest[(ren * step) + (col * channels) + RED] =  (unsigned char) (avg);
	dest[(ren * step) + (col * channels) + GREEN] = (unsigned char) (avg);
	dest[(ren * step) + (col * channels) + BLUE] = (unsigned char) (avg);
}

int main(int argc, char* argv[]) {
	int step, size;
	unsigned char *dev_src, *dev_dest;

	char src_name[255];
	char filter_t[255];
	char dest_name[255];

	strcpy(src_name, argv[1]);
	strcpy(filter_t, argv[2]);
	strcpy(dest_name, argv[3]);

	IplImage *src = cvLoadImage(src_name, CV_LOAD_IMAGE_COLOR);
	IplImage *dest = cvCreateImage(cvSize(src->width, src->height), IPL_DEPTH_8U, 3);

	size = src->width * src->height * src->nChannels * sizeof(uchar);
	step = src->widthStep / sizeof(uchar);

	hipMalloc((void**) &dev_src, size);
	hipMalloc((void**) &dev_dest, size);

	hipMemcpy(dev_src, src->imageData, size, hipMemcpyHostToDevice);

 
    gray<<<src->height, src->width>>>(dev_src, dev_dest, src->width, src->height, step, src->nChannels);
 

	hipMemcpy(dest->imageData, dev_dest, size, hipMemcpyDeviceToHost);

	hipFree(dev_dest);
	hipFree(dev_src);

	return 0;
}
